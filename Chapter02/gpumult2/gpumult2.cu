#include "hip/hip_runtime.h"
// example 2.15 gpumult1 GPU simple matrix multiply one thread per output element with restrict
// introduces lambda function for 2D arressing

#include "cx.h"
#include "cxtimers.h"
#include <random>

__global__ void gpumult2(r_Ptr<float> C,cr_Ptr<float> A,cr_Ptr<float> B,int Ay,int Ax,int Bx)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;  // col index j
	int i = blockIdx.y*blockDim.y + threadIdx.y;  // row index i
	if(j >= Ay || i >= Bx) return;

	auto idx = [&Bx](int i,int j){ return i*Bx+j; }; // lambda function

	// this the the closest we can get to using convential mathmatical notation
	// while retaining the speed advantage of restrict.
	C[idx(i,j)] = 0.0;
	for(int k=0;k<Ax;k++) C[idx(i,j)] += A[idx(i,k)]*B[idx(k,j)];
}

int main(int argc,char *argv[])
{
	int nacc = (argc > 1) ? atoi(argv[1]) : 100;
	int Arow = (argc > 2) ? atoi(argv[2]) : 1024; // default 2^10
	int Acol = (argc > 3) ? atoi(argv[3]) : Arow;
	int Brow = Acol;
	int Bcol = (argc > 4) ? atoi(argv[4]) : Brow;
	int Crow = Arow;
	int Ccol = Bcol;

	uint tilex = (argc > 5) ? atoi(argv[5]) : 32;  // thread-block x
	uint tiley = (argc > 6) ? atoi(argv[6]) : 8;   // thread-block y

	thrust::host_vector<float>       A(Arow*Acol);
	thrust::host_vector<float>       B(Brow*Bcol);
	thrust::host_vector<float>       C(Crow*Ccol);
	thrust::device_vector<float> dev_C(Crow*Ccol);
	thrust::device_vector<float> dev_A(Arow*Acol);
	thrust::device_vector<float> dev_B(Brow*Bcol);

	// initialise A and B with random numbers
	std::default_random_engine gen(12345678);
	std::uniform_real_distribution<float> fran(0.0,1.0);
	for(int k = 0; k<Arow*Acol; k++) A[k] = fran(gen);
	for(int k = 0; k<Brow*Bcol; k++) B[k] = fran(gen);

	dev_A = A;  // H2D copy
	dev_B = B;  // H2D copy
	dim3 threads ={tilex,tiley,1};
	dim3 blocks ={(Bcol+threads.x-1)/threads.x,(Arow+threads.y-1)/threads.y,1};

	cx::timer tim;
	for(int k=0;k<nacc;k++){
		gpumult2<<<blocks,threads>>>(dev_C.data().get(),dev_A.data().get(),dev_B.data().get(),Arow,Acol,Bcol);
	}
	hipDeviceSynchronize();  // wait for kernel
	double t2 = tim.lap_ms()/(double)(nacc);

	C = dev_C;               // D2H copy
	double flops = 2.0*(double)Arow*(double)Acol*(double)Bcol;
	double gflops = flops/(t2*1000000.0);
	double gbytes = gflops*6.0; // i.e 12 bytes per term
	printf("A %d x %d B %d x %d gpu time %.3f ms GFlops %.3f GBytes %.3f\n",
		Arow,Acol,Brow,Bcol,t2,gflops,gbytes);
	return 0;
}
