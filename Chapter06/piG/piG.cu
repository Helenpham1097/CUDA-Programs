#include "hip/hip_runtime.h"
// example 6.7 piG cuRand Device API 

#include "cx.h"  
#include "cxtimers.h"
#include "hiprand/hiprand_kernel.h"
#include <random>
template <typename S> __global__ void

init_generator(long long seed,S *states)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init(seed+id,0,0,&states[id]); // faster
	//hiprand_init(seed, id, 0, &states[id]); // statistically better
}

template <typename S> __global__ void piG(float *tsum,S *states,int points)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	S state = states[id]; // restore state from previous call
	float sum = 0.0f;
	for(int i = 0; i < points; i++) {
		float x = hiprand_uniform(&state);
		float y = hiprand_uniform(&state);
		if(x*x + y*y < 1.0f) sum++; // point inside circle?
	}
	tsum[id] += sum;
	states[id] = state;  // save state for next call
}

int main(int argc,char *argv[])
{
	std::random_device rd;
	int shift =      (argc > 1) ? atoi(argv[1]) : 18;
	long long seed = (argc > 2) ? atoll(argv[2]) : rd();
	int blocks =     (argc > 3) ? atoi(argv[3]) : 2048;
	int threads =    (argc > 4) ? atoi(argv[4]) : 1024;
	long long ntot = (long long)1 << shift;

	int size = threads*blocks;
	int nthread = (ntot+size-1)/size;
	ntot = (long long)nthread*size;

	thrust::device_vector<float> tsum(size);         // thread sums
	thrust::device_vector <hiprandState> state(size); // generator states

	cx::timer tim;   // start clock
	init_generator<<<blocks,threads>>>(seed, state.data().get());
	piG<<<blocks,threads>>>(tsum.data().get(), state.data().get(), nthread);
	double sum_inside = thrust::reduce(tsum.begin(),tsum.end());
	double t1 = tim.lap_ms(); // record time

	double pi = 4.0*sum_inside/(double)ntot;
	double frac_error = 1000000.0*(pi - cx::pi<double>)/cx::pi<double>; // ppm
	printf("piG = %10.8f err %.3f, ntot %lld, time %.3f ms\n", pi,frac_error, ntot, t1);
	return 0;
}
